#include <hip/hip_runtime.h>
#include "union_find.hxx"
#include "time_measure_util.h"
#include <algorithm>
#include <cstdlib>
#include "ECLgraph.h"
#include <thrust/transform_scan.h>
#include <thrust/transform.h>
#include "maximum_matching_vertex_based.h"
#include "multicut_solver_options.h"
#include "dual_solver.h"
#include "edge_contractions_woc.h"
#include "rama_utils.h"

struct is_negative
{
    __host__ __device__
        bool operator()(const float x)
        {
            return x < 0.0;
        }
};
bool has_bad_contractions(const dCOO& A)
{
    const thrust::device_vector<float> d = A.diagonal();
    return thrust::count_if(d.begin(), d.end(), is_negative()) > 0;
}

std::tuple<thrust::device_vector<int>, int> contraction_mapping_by_maximum_matching(dCOO& A, const float mean_multiplier_mm)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;
    thrust::device_vector<int> node_mapping;
    int nr_matched_edges;
    std::tie(node_mapping, nr_matched_edges) = filter_edges_by_matching_vertex_based(A.export_undirected(), mean_multiplier_mm);
    return {compress_label_sequence(node_mapping, node_mapping.size() - 1), nr_matched_edges};
}

std::tuple<std::vector<int>, double, std::vector<std::vector<int>> > rama_cuda(dCOO& A, const multicut_solver_options& opts)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    assert(A.is_directed());

    const double final_lb = dual_solver(A, opts.max_cycle_length_lb, opts.num_dual_itr_lb, opts.tri_memory_factor, opts.num_outer_itr_dual);

    const double initial_lb = A.sum();
    std::cout << "initial energy = " << initial_lb << "\n";

    thrust::device_vector<int> node_mapping(A.max_dim());
    thrust::sequence(node_mapping.begin(), node_mapping.end());

    std::vector<std::vector<int>> timeline;

    if (opts.only_compute_lb)
        return {std::vector<int>(), final_lb, timeline};
        
    bool try_edges_to_contract_by_maximum_matching = true;
    if (opts.matching_thresh_crossover_ratio > 1.0)
        try_edges_to_contract_by_maximum_matching = false;
    
    for(size_t iter=0; A.nnz() > 0; ++iter)
    {
        if (iter > 0)
        {
            dual_solver(A, opts.max_cycle_length_primal, opts.num_dual_itr_primal, 1.0, 1);
        }
        thrust::device_vector<int> cur_node_mapping;
        int nr_edges_to_contract;
        if(try_edges_to_contract_by_maximum_matching)
        {
            std::tie(cur_node_mapping, nr_edges_to_contract) = contraction_mapping_by_maximum_matching(A, opts.mean_multiplier_mm);
            if(nr_edges_to_contract < A.rows() * opts.matching_thresh_crossover_ratio)
            {
                std::cout << "# edges to contract = " << nr_edges_to_contract << ", # vertices = " << A.rows() << "\n";
                std::cout << "switching to MST based contraction edge selection\n";
                try_edges_to_contract_by_maximum_matching = false;    
            }
        }
        else
        {
            edge_contractions_woc c_mapper(A);
            std::tie(cur_node_mapping, nr_edges_to_contract) = c_mapper.find_contraction_mapping();
        }

        if(nr_edges_to_contract == 0)
        {
            std::cout << "# iterations = " << iter << "\n";
            break;
        }

        dCOO new_A = A.contract_cuda(cur_node_mapping);
        std::cout << "original A size " << A.cols() << "x" << A.rows() << "\n";
        std::cout << "contracted A size " << new_A.cols() << "x" << new_A.rows() << "\n";
        assert(new_A.cols() < A.cols());

        const thrust::device_vector<float> diagonal = new_A.diagonal();
        const float energy_reduction = thrust::reduce(diagonal.begin(), diagonal.end());
        std::cout << "energy reduction " << energy_reduction << "\n";
        if(has_bad_contractions(new_A))
            throw std::runtime_error("Found bad contractions");

        thrust::swap(A,new_A);
        A.remove_diagonal();
        std::cout << "energy after iteration " << iter << ": " << A.sum() << ", #components = " << A.cols() << "\n";
        thrust::gather(node_mapping.begin(), node_mapping.end(), cur_node_mapping.begin(), node_mapping.begin());
        if (opts.dump_timeline)
        {
            std::vector<int> current_timeline(node_mapping.size());
            thrust::copy(node_mapping.begin(), node_mapping.end(), current_timeline.begin());
            timeline.push_back(current_timeline);
        }
        if (opts.max_time_sec >= 0)
        {
            std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
            auto time = std::chrono::duration_cast<std::chrono::seconds>(end - begin).count();
            if (time > opts.max_time_sec)
                break;
        }
    }

    const double lb = A.sum();
    std::cout << "final energy = " << lb << "\n";

    std::vector<int> h_node_mapping(node_mapping.size());
    thrust::copy(node_mapping.begin(), node_mapping.end(), h_node_mapping.begin());
    return {h_node_mapping, final_lb, timeline};
}

std::tuple<std::vector<int>, double, int, std::vector<std::vector<int>> > rama_cuda(const std::vector<int>& i, const std::vector<int>& j, const std::vector<float>& costs, const multicut_solver_options& opts)
{
    const int cuda_device = get_cuda_device();
    hipSetDevice(cuda_device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device);
    std::cout << "Going to use " << prop.name << " " << prop.major << "." << prop.minor << ", device number " << cuda_device << "\n";

    dCOO A(i.begin(), i.end(), j.begin(), j.end(), costs.begin(), costs.end(), true);
    
    std::vector<int> h_node_mapping;
    double lb;
    std::vector<std::vector<int>> timeline;
    
    std::chrono::steady_clock::time_point start_time = std::chrono::steady_clock::now();
    std::tie(h_node_mapping, lb, timeline) = rama_cuda(A, opts);
    std::chrono::steady_clock::time_point end_time = std::chrono::steady_clock::now();
    int time_duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
    return {h_node_mapping, lb, time_duration, timeline};
}
