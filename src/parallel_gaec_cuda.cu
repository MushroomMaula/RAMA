#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "dCOO.h"
#include "union_find.hxx"
#include "time_measure_util.h"
#include <algorithm>
#include <cstdlib>
#include "ECLgraph.h"
#include <thrust/transform_scan.h>
#include <thrust/transform.h>
#include "maximum_matching_vertex_based.h"
#include "icp_small_cycles.h"
#include "parallel_gaec_utils.h"

thrust::device_vector<int> compress_label_sequence(const thrust::device_vector<int>& data, const int max_label)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;

    assert(*thrust::max_element(data.begin(), data.end()) <= max_label);

    // first get mask of used labels
    thrust::device_vector<int> label_mask(max_label + 1, 0);
    thrust::scatter(thrust::constant_iterator<int>(1), thrust::constant_iterator<int>(1) + data.size(), data.begin(), label_mask.begin());

    // get map of original labels to consecutive ones
    thrust::device_vector<int> label_to_consecutive(max_label + 1);
    thrust::exclusive_scan(label_mask.begin(), label_mask.end(), label_to_consecutive.begin());

    // apply compressed label map
    thrust::device_vector<int> result(data.size(), 0);
    thrust::gather(data.begin(), data.end(), label_to_consecutive.begin(), result.begin());

    return result;
}


thrust::device_vector<float> per_cc_cost(const dCOO& A, const dCOO& C, const thrust::device_vector<int>& node_mapping, const int nr_ccs)
{
    thrust::device_vector<float> d = A.diagonal();
    return d;
}

struct is_negative
{
    __host__ __device__
        bool operator()(const float x)
        {
            return x < 0.0;
        }
};
bool has_bad_contractions(const dCOO& A)
{
    const thrust::device_vector<float> d = A.diagonal();
    return thrust::count_if(d.begin(), d.end(), is_negative()) > 0;
}

struct remove_bad_contraction_edges_func
{
    const int nr_ccs;
    const float* cc_cost;
    __host__ __device__
        int operator()(thrust::tuple<int,int> t)
        {
            const int cc = thrust::get<0>(t);
            const int node_id = thrust::get<1>(t);
            if (cc_cost[cc] > 0.0)
                return cc;
            return node_id + nr_ccs;
        }
};

thrust::device_vector<int> discard_bad_contractions(const dCOO& contracted_A, const thrust::device_vector<int>& node_mapping)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;
    int nr_ccs = *thrust::max_element(node_mapping.begin(), node_mapping.end()) + 1;
    // for each component, how profitable was it to contract it?
    const thrust::device_vector<float> d = contracted_A.diagonal();

    remove_bad_contraction_edges_func func({nr_ccs, thrust::raw_pointer_cast(d.data())}); 

    thrust::device_vector<int> good_node_mapping = node_mapping;
    thrust::device_vector<int> input_node_ids(node_mapping.size());
    thrust::sequence(input_node_ids.begin(), input_node_ids.end(), 0);

    auto first = thrust::make_zip_iterator(thrust::make_tuple(good_node_mapping.begin(), input_node_ids.begin()));
    auto last = thrust::make_zip_iterator(thrust::make_tuple(good_node_mapping.end(), input_node_ids.end()));
    thrust::transform(first, last, good_node_mapping.begin(), func);
    
    good_node_mapping = compress_label_sequence(good_node_mapping, nr_ccs + node_mapping.size());
    assert(*thrust::max_element(good_node_mapping.begin(), good_node_mapping.end()) > nr_ccs);
    return good_node_mapping;
}


struct negative_edge_indicator_func
{
    const float w = 0.0;
    __host__ __device__
        bool operator()(const thrust::tuple<int,int,float> t)
        {
            if(thrust::get<2>(t) <= w)
                return true;
            return false;
        }
};

struct edge_comparator_func {
    __host__ __device__
        inline bool operator()(const thrust::tuple<int, int, float>& a, const thrust::tuple<int, int, float>& b)
        {
            return thrust::get<2>(a) > thrust::get<2>(b);
        } 
};

std::tuple<thrust::device_vector<int>, int> contraction_mapping_by_sorting(dCOO& A, const float retain_ratio)
{
    assert(A.is_directed());
    thrust::device_vector<int> row_ids = A.get_row_ids();
    thrust::device_vector<int> col_ids = A.get_col_ids();
    thrust::device_vector<float> data = A.get_data();

    auto first = thrust::make_zip_iterator(thrust::make_tuple(col_ids.begin(), row_ids.begin(), data.begin()));
    auto last = thrust::make_zip_iterator(thrust::make_tuple(col_ids.end(), row_ids.end(), data.end()));

    const double smallest_edge_weight = *thrust::min_element(data.begin(), data.end());
    const double largest_edge_weight = *thrust::max_element(data.begin(), data.end());
    const float mid_edge_weight = retain_ratio * largest_edge_weight;

    auto new_last = thrust::remove_if(first, last, negative_edge_indicator_func({mid_edge_weight}));
    const size_t nr_remaining_edges = std::distance(first, new_last);
    col_ids.resize(nr_remaining_edges);
    row_ids.resize(nr_remaining_edges);
    if (nr_remaining_edges == 0)
        return {thrust::device_vector<int>(0), 0};

    /*
    if(max_contractions < nr_positive_edges)
    {
        auto first = thrust::make_zip_iterator(thrust::make_tuple(col_ids.begin(), row_ids.begin(), data.begin()));
        auto last = thrust::make_zip_iterator(thrust::make_tuple(col_ids.end(), row_ids.end(), data.end()));
        thrust::sort(first, last, edge_comparator_func()); // TODO: faster through sort by keys?

        col_ids.resize(max_contractions);
        row_ids.resize(max_contractions);
        data.resize(max_contractions);
    }
    */

    // add reverse edges
    std::tie(row_ids, col_ids) = to_undirected(row_ids.begin(), row_ids.end(), col_ids.begin(), col_ids.end());

    assert(col_ids.size() == row_ids.size());
    coo_sorting(row_ids, col_ids);
    thrust::device_vector<int> row_offsets = compute_offsets(row_ids, A.max_dim() - 1);

    thrust::device_vector<int> cc_labels(A.max_dim());
    computeCC_gpu(A.max_dim(), col_ids.size(), 
            thrust::raw_pointer_cast(row_offsets.data()), thrust::raw_pointer_cast(col_ids.data()), 
            thrust::raw_pointer_cast(cc_labels.data()), get_cuda_device());

    thrust::device_vector<int> node_mapping = compress_label_sequence(cc_labels, cc_labels.size() - 1);
    const int nr_ccs = *thrust::max_element(node_mapping.begin(), node_mapping.end()) + 1;

    assert(nr_ccs < A.max_dim());

    return {node_mapping, row_ids.size()};

}

std::tuple<thrust::device_vector<int>, int> contraction_mapping_by_maximum_matching(dCOO& A)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;
    MEASURE_FUNCTION_EXECUTION_TIME;
    thrust::device_vector<int> node_mapping;
    int nr_matched_edges;
    std::tie(node_mapping, nr_matched_edges) = filter_edges_by_matching_vertex_based(A.export_undirected());
    return {compress_label_sequence(node_mapping, node_mapping.size() - 1), nr_matched_edges};
}

std::vector<int> parallel_gaec_cuda(dCOO& A)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;
    assert(A.is_directed());

    const double initial_lb = A.sum();
    std::cout << "initial energy = " << initial_lb << "\n";

    thrust::device_vector<int> node_mapping(A.rows());
    thrust::sequence(node_mapping.begin(), node_mapping.end());
    double contract_ratio = 0.5;

    bool try_edges_to_contract_by_maximum_matching = true;
    
    for(size_t iter=0;; ++iter)
    {
        //const size_t nr_edges_to_contract = std::max(size_t(1), size_t(A.rows() * contract_ratio));
        // if (iter > 0)
        // {
        //     dCOO A_dir = A.export_directed();
        //     parallel_small_cycle_packing_cuda(A_dir, 1, 0);
        //     A = A_dir.export_undirected(); //TODO: just replace data ?
        // }
        thrust::device_vector<int> cur_node_mapping;
        int nr_edges_to_contract;
        if(try_edges_to_contract_by_maximum_matching)
        {
            std::tie(cur_node_mapping, nr_edges_to_contract) = contraction_mapping_by_maximum_matching(A);
            if(nr_edges_to_contract < A.rows()*0.1)
            {
                std::cout << "# edges to contract = " << nr_edges_to_contract << ", # vertices = " << A.rows() << "\n";
                std::cout << "switching to sorting based contraction edge selection\n";
                try_edges_to_contract_by_maximum_matching = false;    
            }
        }
        if(!try_edges_to_contract_by_maximum_matching)
            std::tie(cur_node_mapping, nr_edges_to_contract) = contraction_mapping_by_sorting(A, contract_ratio);


        //std::cout << "iter " << iter << ", edge contraction ratio = " << contract_ratio << ", # edges to contract request " << nr_edges_to_contract << ", # nr edges to contract provided = " << contract_cols.size() << "\n";

        if(nr_edges_to_contract == 0)
        {
            std::cout << "# iterations = " << iter << "\n";
            break;
        }

        dCOO new_A = A.contract_cuda(cur_node_mapping);
        std::cout << "original A size " << A.cols() << "x" << A.rows() << "\n";
        std::cout << "contracted A size " << new_A.cols() << "x" << new_A.rows() << "\n";
        assert(new_A.cols() < A.cols());

        const thrust::device_vector<float> diagonal = new_A.diagonal();
        const float energy_reduction = thrust::reduce(diagonal.begin(), diagonal.end());
        std::cout << "energy reduction " << energy_reduction << "\n";
        //if(energy_reduction < 0.0)
        if(has_bad_contractions(new_A))
        {
            if(!try_edges_to_contract_by_maximum_matching)
                contract_ratio *= 2.0; 
            //contract_ratio = std::max(contract_ratio, 0.005);
            // get contraction edges of the components which
            int nr_ccs = *thrust::max_element(cur_node_mapping.begin(), cur_node_mapping.end()) + 1;
            cur_node_mapping = discard_bad_contractions(new_A, cur_node_mapping);
            int good_nr_ccs = *thrust::max_element(cur_node_mapping.begin(), cur_node_mapping.end()) + 1;
            assert(good_nr_ccs > nr_ccs);
            std::cout << "Reverted from " << nr_ccs << " connected components to " << good_nr_ccs << "\n";
            if (good_nr_ccs == cur_node_mapping.size()) 
                break;
            
            new_A = A.contract_cuda(cur_node_mapping);
            assert(!has_bad_contractions(new_A));
        }
        else
        {
            if(!try_edges_to_contract_by_maximum_matching)
            {
                contract_ratio *= 0.5;//1.3;
                contract_ratio = std::min(contract_ratio, 0.35);
            }
        }

        thrust::swap(A,new_A);
        A.remove_diagonal();
        std::cout << "energy after iteration " << iter << ": " << A.sum() << ", #components = " << A.cols() << "\n";
        thrust::gather(node_mapping.begin(), node_mapping.end(), cur_node_mapping.begin(), node_mapping.begin());
    }

    const double lb = A.sum();
    std::cout << "final energy = " << lb << "\n";

    std::vector<int> h_node_mapping(node_mapping.size());
    thrust::copy(node_mapping.begin(), node_mapping.end(), h_node_mapping.begin());
    return h_node_mapping;
}

void print_obj_original(const std::vector<int>& h_node_mapping, const std::vector<int>& i, const std::vector<int>& j, const std::vector<float>& costs)
{
    double obj = 0;
    const int nr_edges = costs.size();
    for (int e = 0; e < nr_edges; e++)
    {
        const int e1 = i[e];
        const int e2 = j[e];
        const float c = costs[e];
        if (h_node_mapping[e1] != h_node_mapping[e2])
            obj += c;
    }
    std::cout<<"Cost w.r.t original objective: "<<obj<<std::endl;
}

std::vector<int> parallel_gaec_cuda(const std::vector<int>& i, const std::vector<int>& j, const std::vector<float>& costs)
{
    const int cuda_device = get_cuda_device();
    hipSetDevice(cuda_device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device);
    std::cout << "Going to use " << prop.name << " " << prop.major << "." << prop.minor << ", device number " << cuda_device << "\n";

    dCOO A(i.begin(), i.end(), j.begin(), j.end(), costs.begin(), costs.end(), true);

    const std::vector<int> h_node_mapping = parallel_gaec_cuda(A);
    print_obj_original(h_node_mapping, i, j, costs); 
    
    return h_node_mapping;
}

std::vector<int> parallel_gaec_cuda(thrust::device_vector<int>&& i, thrust::device_vector<int>&& j, thrust::device_vector<float>&& costs)
{
    dCOO A(std::move(i), std::move(j), std::move(costs), true);
    const std::vector<int> h_node_mapping = parallel_gaec_cuda(A);
    
    return h_node_mapping;

}
