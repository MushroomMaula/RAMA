#include "dCSR.h"
#include <thrust/transform.h>
#include <thrust/tuple.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>

void dCSR::print() const
{
    assert(rows() == row_offsets.size()-1);
    assert(col_ids.size() == data.size());
    std::cout << "dimension = " << rows() << "," << cols() << "\n";
    for(size_t i=0; i<rows(); ++i)
        for(size_t l=row_offsets[i]; l<row_offsets[i+1]; ++l)
            std::cout << i << "," << col_ids[l] << "," << data[l] << "\n"; 
}

dCSR dCSR::transpose(hipsparseHandle_t handle)
{
    dCSR t;
    t.cols_ = rows();
    t.rows_ = cols();

    std::cout << "t.row_offsets.size() = " << t.row_offsets.size() << "\n";
    t.row_offsets = thrust::device_vector<int>(cols()+1);
    std::cout << "t.row_offsets.size() after = " << t.row_offsets.size() << "\n";
    t.col_ids = thrust::device_vector<int>(nnz());
    t.data = thrust::device_vector<float>(nnz());

    checkCuSparseError(hipsparseScsr2csc(handle, rows(), cols(), nnz(), 
			thrust::raw_pointer_cast(data.data()), thrust::raw_pointer_cast(row_offsets.data()), thrust::raw_pointer_cast(col_ids.data()),
			thrust::raw_pointer_cast(t.data.data()), thrust::raw_pointer_cast(t.col_ids.data()), thrust::raw_pointer_cast(t.row_offsets.data()),
            HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO),
            "transpose failed");

    return t;
}

dCSR multiply(hipsparseHandle_t handle, const dCSR& A, const dCSR& B)
{
    assert(A.cols() == B.rows());
    int nnzC;
    int *nnzTotalDevHostPtr = &nnzC;
    float duration;
    dCSR C;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipsparseMatDescr_t descrA;
    hipsparseMatDescr_t descrB;
    hipsparseMatDescr_t descrC;

    checkCuSparseError(hipsparseCreateMatDescr(&descrA), "Matrix descriptor init failed");
    checkCuSparseError(hipsparseCreateMatDescr(&descrB), "Matrix descriptor init failed");
    checkCuSparseError(hipsparseCreateMatDescr(&descrC), "Matrix descriptor init failed");
    checkCuSparseError(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL), "hipsparseSetMatType failed");
    checkCuSparseError(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO), "hipsparseSetMatIndexBase failed");
    checkCuSparseError(hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL), "hipsparseSetMatType failed");
    checkCuSparseError(hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO), "hipsparseSetMatIndexBase failed");
    checkCuSparseError(hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL), "hipsparseSetMatType failed");
    checkCuSparseError(hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO), "hipsparseSetMatIndexBase failed");

    // ############################
    hipEventRecord(start);
    // ############################

    // Allocate memory for row indices
    C.row_offsets = thrust::device_vector<int>(A.rows()+1);

    // Precompute number of nnz in C
    checkCuSparseError(hipsparseXcsrgemmNnz(
                handle,
                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                A.rows(), B.cols(), A.cols(),
                descrA, A.nnz(), thrust::raw_pointer_cast(A.row_offsets.data()), thrust::raw_pointer_cast(A.col_ids.data()),
                descrB, B.nnz(), thrust::raw_pointer_cast(B.row_offsets.data()), thrust::raw_pointer_cast(B.col_ids.data()),
                descrC, thrust::raw_pointer_cast(C.row_offsets.data()), nnzTotalDevHostPtr), "cuSparse: Precompute failed"
            );

    C.rows_ = A.rows();
    C.cols_ = B.cols();
    C.col_ids = thrust::device_vector<int>(nnzC);
    C.data = thrust::device_vector<float>(nnzC);

    // Compute SpGEMM
    checkCuSparseError(hipsparseScsrgemm(
                handle,
                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                A.rows(), B.cols(), A.cols(),
                descrA, A.nnz(), thrust::raw_pointer_cast(A.data.data()), thrust::raw_pointer_cast(A.row_offsets.data()), thrust::raw_pointer_cast(A.col_ids.data()),
                descrB, B.nnz(), thrust::raw_pointer_cast(B.data.data()), thrust::raw_pointer_cast(B.row_offsets.data()), thrust::raw_pointer_cast(B.col_ids.data()),
                descrC, thrust::raw_pointer_cast(C.data.data()), thrust::raw_pointer_cast(C.row_offsets.data()), thrust::raw_pointer_cast(C.col_ids.data())),
            "cuSparse: SpGEMM failed");

    // ############################
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    // ############################

    hipEventElapsedTime(&duration, start, stop);

    checkCuSparseError(hipsparseDestroyMatDescr(descrA), "Matrix descriptor destruction failed");
    checkCuSparseError(hipsparseDestroyMatDescr(descrB), "Matrix descriptor destruction failed");
    checkCuSparseError(hipsparseDestroyMatDescr(descrC), "Matrix descriptor destruction failed");

    return C;
}

std::tuple<thrust::host_vector<int>, thrust::host_vector<int>, thrust::host_vector<float>> dCSR::export_coo(hipsparseHandle_t handle)
{
    thrust::host_vector<int> h_col_ids(col_ids);
    thrust::host_vector<float> h_data(data);
    thrust::device_vector<int> row_ids(nnz());

    hipsparseXcsr2coo(handle, thrust::raw_pointer_cast(row_offsets.data()), nnz(), cols(), thrust::raw_pointer_cast(row_ids.data()), HIPSPARSE_INDEX_BASE_ZERO);
            
    thrust::host_vector<int> h_row_ids(row_ids);

    return {h_col_ids, h_row_ids, h_data}; 
}

thrust::device_vector<int> dCSR::row_ids(hipsparseHandle_t handle) const
{
    thrust::device_vector<int> _row_ids(nnz());

    hipsparseXcsr2coo(handle, thrust::raw_pointer_cast(row_offsets.data()), nnz(), cols(), thrust::raw_pointer_cast(_row_ids.data()), HIPSPARSE_INDEX_BASE_ZERO);
            
    return _row_ids;
}

struct diag_to_zero_func
{
    __host__ __device__
        void operator()(thrust::tuple<int&,int&,float&> t)
        {
            if(thrust::get<0>(t) == thrust::get<1>(t))
                thrust::get<2>(t) = 0.0;
        }
};
void dCSR::set_diagonal_to_zero(hipsparseHandle_t handle)
{
    thrust::device_vector<int> _row_ids = row_ids(handle);
    
     auto begin = thrust::make_zip_iterator(thrust::make_tuple(col_ids.begin(), _row_ids.begin(), data.begin()));
     auto end = thrust::make_zip_iterator(thrust::make_tuple(col_ids.end(), _row_ids.end(), data.end()));


     thrust::for_each(thrust::device, begin, end, diag_to_zero_func());
}

float dCSR::sum()
{
    return thrust::reduce(data.begin(), data.end(), (float) 0.0, thrust::plus<float>());
}

