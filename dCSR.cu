#include "hip/hip_runtime.h"
#include "dCSR.h"
#include <thrust/transform.h>
#include <thrust/tuple.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <ECLgraph.h>
#include "time_measure_util.h"

void dCSR::print() const
{
    assert(rows() == row_offsets.size()-1);
    assert(col_ids.size() == data.size());
    std::cout << "dimension = " << rows() << "," << cols() << "\n";
    for(size_t i=0; i<rows(); ++i)
        for(size_t l=row_offsets[i]; l<row_offsets[i+1]; ++l)
            std::cout << i << "," << col_ids[l] << "," << data[l] << "\n"; 
}

dCSR dCSR::transpose(hipsparseHandle_t handle)
{
    MEASURE_FUNCTION_EXECUTION_TIME
    dCSR t;
    t.cols_ = rows();
    t.rows_ = cols();

    t.row_offsets = thrust::device_vector<int>(cols()+1);
    t.col_ids = thrust::device_vector<int>(nnz());
    t.data = thrust::device_vector<float>(nnz());

    // make buffer
    void* dbuffer = NULL;
    size_t bufferSize = 0;
    checkCuSparseError(hipsparseCsr2cscEx2_bufferSize(handle, rows(), cols(), nnz(), 
			thrust::raw_pointer_cast(data.data()), thrust::raw_pointer_cast(row_offsets.data()), thrust::raw_pointer_cast(col_ids.data()),
			thrust::raw_pointer_cast(t.data.data()), thrust::raw_pointer_cast(t.row_offsets.data()), thrust::raw_pointer_cast(t.col_ids.data()), 
            HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &bufferSize), "transpose buffer failed");
    
    checkCudaError(hipMalloc((void**) &dbuffer, bufferSize), "transpose buffer allocation failed");

    checkCuSparseError(hipsparseCsr2cscEx2(handle, rows(), cols(), nnz(), 
			thrust::raw_pointer_cast(data.data()), thrust::raw_pointer_cast(row_offsets.data()), thrust::raw_pointer_cast(col_ids.data()),
			thrust::raw_pointer_cast(t.data.data()), thrust::raw_pointer_cast(t.row_offsets.data()), thrust::raw_pointer_cast(t.col_ids.data()), 
            HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, dbuffer),
            "transpose failed");

    hipFree(dbuffer);
    return t;
}

template <typename T>
struct non_zero_indicator_func
{
    const T _tol;
    non_zero_indicator_func(T tol): _tol(tol) {} 

    __host__ __device__
        bool operator()(const thrust::tuple<int,int,float> t)
        {
            if(fabs(thrust::get<2>(t)) >= _tol)
                return true;
            else
                return false;
        }
};

void dCSR::compress(hipsparseHandle_t handle, const float tol)
{
    MEASURE_FUNCTION_EXECUTION_TIME
    thrust::device_vector<int> _row_ids = row_ids(handle);
    
    auto first = thrust::make_zip_iterator(thrust::make_tuple(col_ids.begin(), _row_ids.begin(), data.begin()));
    auto last = thrust::make_zip_iterator(thrust::make_tuple(col_ids.end(), _row_ids.end(), data.end()));

    auto new_last = thrust::remove_if(first, last, non_zero_indicator_func<float>(tol));

    const size_t nr_non_zeros = std::distance(first, new_last);
    col_ids.resize(nr_non_zeros);
    _row_ids.resize(nr_non_zeros);
    data.resize(nr_non_zeros);

    coo_sorting(col_ids, _row_ids, data);

    // now row indices are non-decreasing
    assert(thrust::is_sorted(_row_ids.begin(), _row_ids.end()));

    cols_ = *thrust::max_element(col_ids.begin(), col_ids.end()) + 1;
    rows_ = _row_ids.back() + 1;

    row_offsets = thrust::device_vector<int>(rows_ + 1);
    hipsparseXcoo2csr(handle, thrust::raw_pointer_cast(_row_ids.data()), nnz(), rows(), thrust::raw_pointer_cast(row_offsets.data()), HIPSPARSE_INDEX_BASE_ZERO);
}

// // Inspired from: https://docs.nvidia.com/cuda/cusparse/index.html#csr2csr_compress
// dCSR dCSR::compress(hipsparseHandle_t handle, const float tol) const
// {
//     MEASURE_FUNCTION_EXECUTION_TIME
//     thrust::device_vector<int> nnz_per_row_interm = thrust::device_vector<int>(rows(), 0);
//     thrust::device_vector<int> total_nnz_interm = thrust::device_vector<int>(1);

//     hipsparseMatDescr_t descrA;
//     checkCuSparseError(hipsparseCreateMatDescr(&descrA), "Matrix descriptor init failed");
//     checkCuSparseError(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO), "cusparseSetMatIndex failed");
//     checkCuSparseError(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL), "hipsparseSetMatType failed");

//     checkCuSparseError(hipsparseSnnz_compress(handle, rows(), descrA, thrust::raw_pointer_cast(data.data()),
//                                          thrust::raw_pointer_cast(row_offsets.data()), thrust::raw_pointer_cast(nnz_per_row_interm.data()),
//                                          thrust::raw_pointer_cast(total_nnz_interm.data()), tol), "cuSparse: stage 1 of compress failed");

//     dCSR c;
//     c.rows_ = rows();

//     c.row_offsets = thrust::device_vector<int>(rows() + 1); 
//     thrust::host_vector<int> total_nnz_interm_h = total_nnz_interm;
//     c.data = thrust::device_vector<float>(total_nnz_interm_h[0]); 
//     c.col_ids = thrust::device_vector<int>(total_nnz_interm_h[0]);

//     //DEBUG:
//     int max_rows = *thrust::max_element(row_offsets.begin(), row_offsets.end());
//     int max_cols = *thrust::max_element(col_ids.begin(), col_ids.end());
//     std::cout<<"max_rows: "<<max_rows<<", max_cols: "<<max_cols<<std::endl;
//     thrust::host_vector<int> row_offsets_h = row_offsets;
//     for (int i = 0; i < rows(); i++)
//     {
//         assert(row_offsets_h[i] >= 0);
//         assert(row_offsets_h[i + 1] >= row_offsets_h[i]);
//     }
//     assert(row_offsets_h[rows()] = nnz());
//     assert(nnz() == col_ids.size());
    
//     checkCuSparseError(hipsparseScsr2csr_compress(handle, rows(), cols(), descrA, 
//                                               thrust::raw_pointer_cast(data.data()),
//                                               thrust::raw_pointer_cast(col_ids.data()), 
//                                               thrust::raw_pointer_cast(row_offsets.data()),
//                                               nnz(), thrust::raw_pointer_cast(nnz_per_row_interm.data()),
//                                               thrust::raw_pointer_cast(c.data.data()), 
//                                               thrust::raw_pointer_cast(c.col_ids.data()),
//                                               thrust::raw_pointer_cast(c.row_offsets.data()), tol), "cuSparse: stage 2 of compress failed");

//     c.cols_ = cols();
//     //DEBUG:
//     int max_rows_after = *thrust::max_element(row_offsets.begin(), row_offsets.end());
//     int max_cols_after = *thrust::max_element(col_ids.begin(), col_ids.end());
//     std::cout<<"max_rows: "<<max_rows_after<<", max_cols: "<<max_cols_after<<std::endl;

//     if (max_cols_after != max_cols or max_rows == 1182577)
//     {
//         for(int i = 54; i < 55; i++)
//         {
//             int j_n = row_offsets[i + 1];
//             int nnz_exp = nnz_per_row_interm[i];
//             for(int j_i = row_offsets[i]; j_i < j_n; j_i++)
//             {
//                 int j = col_ids[j_i];
//                 if (j == max_cols_after)
//                 {
//                     float val = data[j_i];
//                     int a = 1;
//                 }
//             }
//         }
//     }
//     return c;
// }

template <typename T>
struct keep_geq
{
    const T _thresh;
    keep_geq(T thresh): _thresh(thresh) {} 
   __host__ __device__ float operator()(const T &x) const
   {
     return x > _thresh ? x : 0;
   }
};

template <typename T>
struct is_positive
{
    __host__ __device__ bool operator()(const T &x)
    {
        return x > 0;
    }
};

dCSR dCSR::keep_top_k_positive_values(hipsparseHandle_t handle, const int top_k)
{
    MEASURE_FUNCTION_EXECUTION_TIME
    // Create copy of self:
    dCSR p;
    p.rows_ = rows();
    p.cols_ = cols();
    p.row_offsets = row_offsets;
    p.col_ids = col_ids;
    p.data = data;

    // Set all negatives values to zero.
    thrust::transform(p.data.begin(), p.data.end(), p.data.begin(), keep_geq<float>(0.0f));
    int num_positive = thrust::count_if(thrust::device, p.data.begin(), p.data.end(), is_positive<float>());

    if (top_k < num_positive)
    {
        thrust::device_vector<float> temp = p.data;
        thrust::sort(temp.begin(), temp.end(), thrust::greater<float>()); // Ideal would be https://github.com/NVIDIA/thrust/issues/75

        float min_value_to_keep = temp[top_k];
        thrust::transform(p.data.begin(), p.data.end(), p.data.begin(), keep_geq<float>(min_value_to_keep));
    }

    p.compress(handle);

    return p;
}

dCSR multiply(hipsparseHandle_t handle, dCSR& A, dCSR& B)
{
    MEASURE_FUNCTION_EXECUTION_TIME
    assert(A.cols() == B.rows());
    float duration;
    dCSR C;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // CUSPARSE API 
    hipsparseSpMatDescr_t matA, matB, matC;
    float alpha = 1.0f;
    float beta = 0.0f;
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType computeType = HIP_R_32F;
    void* dBuffer1 = NULL, *dBuffer2 = NULL;
    size_t bufferSize1 = 0, bufferSize2 = 0;

    int* rp = thrust::raw_pointer_cast(A.row_offsets.data());

    checkCuSparseError(hipsparseCreateCsr(&matA, A.rows(), A.cols(), A.nnz(),
                                      thrust::raw_pointer_cast(A.row_offsets.data()), 
                                      thrust::raw_pointer_cast(A.col_ids.data()), 
                                      thrust::raw_pointer_cast(A.data.data()),
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F), "Matrix descriptor init failed");

    checkCuSparseError(hipsparseCreateCsr(&matB, B.rows(), B.cols(), B.nnz(),
                                      thrust::raw_pointer_cast(B.row_offsets.data()), 
                                      thrust::raw_pointer_cast(B.col_ids.data()), 
                                      thrust::raw_pointer_cast(B.data.data()),
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F), "Matrix descriptor init failed");

    checkCuSparseError(hipsparseCreateCsr(&matC, A.rows(), B.cols(), 0,
                                      NULL, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F), "Matrix descriptor init failed");

    // SpGEMM Computation
    // ############################
    hipEventRecord(start);
    // ############################
    
    hipsparseSpGEMMDescr_t spgemmDesc;
    checkCuSparseError(hipsparseSpGEMM_createDescr(&spgemmDesc), "sparse MM desc. failed");

    // ask bufferSize1 bytes for external memory
    checkCuSparseError(hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, NULL), "spGEMM work estimation 1 failed");

    checkCudaError(hipMalloc((void**) &dBuffer1, bufferSize1), "buffer 1 allocation failed");

    // inspect the matrices A and B to understand the memory requirement for the next step
    checkCuSparseError(hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, dBuffer1), "spGEMM work estimation 2 failed.");

    // ask bufferSize2 bytes for external memory
    checkCuSparseError(hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, matA, matB, &beta, matC,
                               computeType, HIPSPARSE_SPGEMM_DEFAULT,
                               spgemmDesc, &bufferSize2, NULL), "hipsparseSpGEMM_compute 1 failed");
    checkCudaError(hipMalloc((void**) &dBuffer2, bufferSize2), "buffer 2 allocation failed");

    // compute A * B
    checkCuSparseError(hipsparseSpGEMM_compute(handle, opA, opB,
                                           &alpha, matA, matB, &beta, matC,
                                           computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDesc, &bufferSize2, dBuffer2), "hipsparseSpGEMM_compute 2 failed");
    // get matrix C sizes
    int64_t rows_C, cols_C, nnzC;
    checkCuSparseError(hipsparseSpMatGetSize(matC, &rows_C, &cols_C, &nnzC), "matC get size failed");
    assert(rows_C == A.rows());
    assert(cols_C == B.cols());

    // Allocate memory for C
    C.rows_ = A.rows();
    C.cols_ = B.cols();
    C.row_offsets = thrust::device_vector<int>(A.rows()+1);
    C.col_ids = thrust::device_vector<int>(nnzC);
    C.data = thrust::device_vector<float>(nnzC);

    // update matC with the new pointers
    checkCuSparseError(hipsparseCsrSetPointers(matC, thrust::raw_pointer_cast(C.row_offsets.data()), 
                                                    thrust::raw_pointer_cast(C.col_ids.data()), 
                                                    thrust::raw_pointer_cast(C.data.data())), "Setting matC pointers failed");

    // copy the final products to the matrix C.
    checkCuSparseError(hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc), "Copying to matC failed");

    // ############################
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    // ############################

    hipEventElapsedTime(&duration, start, stop);

    checkCuSparseError(hipsparseSpGEMM_destroyDescr(spgemmDesc), "SPGEMM descriptor destruction failed");
    checkCuSparseError(hipsparseDestroySpMat(matA), "Matrix descriptor destruction failed");
    checkCuSparseError(hipsparseDestroySpMat(matB), "Matrix descriptor destruction failed");
    checkCuSparseError(hipsparseDestroySpMat(matC), "Matrix descriptor destruction failed");
    checkCudaError(hipFree(dBuffer1), "dBuffer1 free failed");
    checkCudaError(hipFree(dBuffer2), "dBuffer2 free failed");

    return C;
}

std::tuple<thrust::host_vector<int>, thrust::host_vector<int>, thrust::host_vector<float>> dCSR::export_coo(hipsparseHandle_t handle)
{
    thrust::host_vector<int> h_col_ids(col_ids);
    thrust::host_vector<float> h_data(data);
    thrust::device_vector<int> row_ids(nnz());

    hipsparseXcsr2coo(handle, thrust::raw_pointer_cast(row_offsets.data()), nnz(), cols(), thrust::raw_pointer_cast(row_ids.data()), HIPSPARSE_INDEX_BASE_ZERO);
            
    thrust::host_vector<int> h_row_ids(row_ids);

    return {h_col_ids, h_row_ids, h_data}; 
}

thrust::device_vector<int> dCSR::row_ids(hipsparseHandle_t handle) const
{
    thrust::device_vector<int> _row_ids(nnz());

    hipsparseXcsr2coo(handle, thrust::raw_pointer_cast(row_offsets.data()), nnz(), cols(), thrust::raw_pointer_cast(_row_ids.data()), HIPSPARSE_INDEX_BASE_ZERO);
            
    return _row_ids;
}

struct diag_to_zero_func
{
    __host__ __device__
        void operator()(thrust::tuple<int&,int&,float&> t)
        {
            if(thrust::get<0>(t) == thrust::get<1>(t))
                thrust::get<2>(t) = 0.0;
        }
};
void dCSR::set_diagonal_to_zero(hipsparseHandle_t handle)
{
    thrust::device_vector<int> _row_ids = row_ids(handle);
    
     auto begin = thrust::make_zip_iterator(thrust::make_tuple(col_ids.begin(), _row_ids.begin(), data.begin()));
     auto end = thrust::make_zip_iterator(thrust::make_tuple(col_ids.end(), _row_ids.end(), data.end()));

     thrust::for_each(thrust::device, begin, end, diag_to_zero_func());
}

float dCSR::sum()
{
    return thrust::reduce(data.begin(), data.end(), (float) 0.0, thrust::plus<float>());
}

thrust::device_vector<int> dCSR::compute_cc(const int device)
{
    thrust::device_vector<int> cc_ids(rows());
    computeCC_gpu(rows(), nnz(), 
                thrust::raw_pointer_cast(row_offsets.data()), 
                thrust::raw_pointer_cast(col_ids.data()), 
                thrust::raw_pointer_cast(cc_ids.data()), device);
    return cc_ids;
}

void dCSR::print_info_of(const int i) const
{   
    std::cout<<"Row offsets of "<<i<<", start: "<<row_offsets[i]<<", end excl.: "<<row_offsets[i+1]<<std::endl;
    std::cout<<"Neighbours:"<<std::endl;
    for(size_t l=row_offsets[i]; l<row_offsets[i+1]; ++l)
        std::cout << i << "," << col_ids[l] << "," << data[l] << "\n"; 
}