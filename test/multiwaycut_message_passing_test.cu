#include "hip/hip_runtime.h"
#include "multiwaycut_message_passing.h"
#include "multiwaycut_text_parser.h"
#include "test.h"

void test_multiway_cut_repulsive_triangle() {
    int nodes = 3;
    int classes = 2;
    std::vector<int> src = {0, 0, 1};
    std::vector<int> dest = {1, 2, 2};
    std::vector<float> edge_costs = {-1.0, -1.0, -1.0};
    std::vector<float> class_costs = {
        1,1, 1,1, 1,1
    };
    thrust::device_vector<int> i;
    thrust::device_vector<int> j;
    thrust::device_vector<float> costs;
    std::tie(i, j, costs) = mwc_to_coo(nodes, classes, class_costs, src, dest, edge_costs);

    thrust::device_vector<int> t1 = std::vector<int>{0, 0, 0, 0, 0, 1, 1};
    thrust::device_vector<int> t2 = std::vector<int>{1, 1, 2, 1, 2, 2, 2};
    thrust::device_vector<int> t3 = std::vector<int>{2, 3, 3, 4, 4, 3, 4};
    dCOO A(i.begin(), i.end(), j.begin(), j.end(), costs.begin(), costs.end(), true);
    multiwaycut_message_passing mwcp(A, nodes, classes, std::move(t1), std::move(t2), std::move(t3));

    const double initial_lb = mwcp.lower_bound();
    std::cout << "initial lb = " << initial_lb << "\n";
    test(std::abs(initial_lb + 3.0) <= 1e-6, "Initial lb before reparametrization must be -3");

    int iterations = 10;
    double last_lb = initial_lb;
    for (int k = 0; k < iterations; ++k) {
        std::cout << "---------------" << "iteration = " << k << "---------------\n";
        mwcp.send_messages_to_triplets();
        double new_lb = mwcp.lower_bound();
        test(new_lb > last_lb || std::abs(new_lb - last_lb) < 1e-6, "Lower bound did not increase after message to triplets");
        last_lb = new_lb;

        mwcp.send_messages_to_edges();
        new_lb = mwcp.lower_bound();
        test(new_lb > last_lb || std::abs(new_lb - last_lb) < 1e-6, "Lower bound did not increase after message to edges");
        last_lb = new_lb;

        mwcp.send_messages_from_sum_to_edges();
        new_lb = mwcp.lower_bound();
        test(new_lb > last_lb || std::abs(new_lb - last_lb) < 1e-6, "Lower bound did not increase after messages from class constraints");
        last_lb = new_lb;
    }

    const double final_lb = mwcp.lower_bound();
    std::cout << "final lb = " << final_lb << "\n";
//    test(std::abs(final_lb - 1.0) <= 1e-6, "Final lb after reparametrization must be -2");
}


void test_multiway_cut_2_nodes_2_classes() {
    int nodes = 2;
    int classes = 2;
    std::vector<int> src = {0};
    std::vector<int> dest = {1};
    std::vector<float> edge_costs = {1.0};
    std::vector<float> class_costs = {
        1,1, 1,1
    };
    thrust::device_vector<int> i;
    thrust::device_vector<int> j;
    thrust::device_vector<float> costs;
    std::tie(i, j, costs) = mwc_to_coo(nodes, classes, class_costs, src, dest, edge_costs);

    thrust::device_vector<int> t1 = std::vector<int>{0, 0};
    thrust::device_vector<int> t2 = std::vector<int>{1, 1};
    thrust::device_vector<int> t3 = std::vector<int>{2, 3};
    dCOO A(i.begin(), i.end(), j.begin(), j.end(), costs.begin(), costs.end(), true);
    multiwaycut_message_passing mwcp(A, nodes, classes, std::move(t1), std::move(t2), std::move(t3));

    const double initial_lb = mwcp.lower_bound();
    std::cout << "initial lb = " << initial_lb << "\n";
    test(std::abs(initial_lb) <= 1e-6, "Initial lb before reparametrization must be 0");

    int iterations = 22;  // Need 21 iterations to reach sufficiently close approximation
    double last_lb = initial_lb;
    for (int k = 0; k < iterations; ++k) {
        std::cout << "---------------" << "iteration=" << k << "---------------\n";
        mwcp.send_messages_to_triplets();
        double new_lb = mwcp.lower_bound();
        test(new_lb > last_lb || std::abs(new_lb - last_lb) < 1e-6, "Lower bound did not increase after message to triplets");
        last_lb = new_lb;

        mwcp.send_messages_to_edges();
        new_lb = mwcp.lower_bound();
        test(new_lb > last_lb || std::abs(new_lb - last_lb) < 1e-6, "Lower bound did not increase after message to edges");
        last_lb = new_lb;

        mwcp.send_messages_from_sum_to_edges();
        new_lb = mwcp.lower_bound();
        test(new_lb > last_lb || std::abs(new_lb - last_lb) < 1e-6, "Lower bound did not increase after messages from class constraints");
        last_lb = new_lb;
    }

    const double final_lb = mwcp.lower_bound();
    std::cout << "final lb = " << final_lb << "\n";
    test(std::abs(final_lb - 2) <= 1e-6, "Final lb after reparametrization must be 2");
}


int main(int argc, char** argv)
{
    std::cout << "Testing repulsive triangle\n";
    test_multiway_cut_repulsive_triangle();
    std::cout << "Testing 2 nodes 2 classes\n";
    test_multiway_cut_2_nodes_2_classes();
}
