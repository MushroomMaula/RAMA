#include "hip/hip_runtime.h"
#include "multiwaycut_message_passing.h"
#include "multiwaycut_text_parser.h"
#include "test.h"

void test_multiway_cut_repulsive_triangle(
    const float edge_cost,
    const std::array<float, 3> c1,  // Class costs for class 1 for all nodes
    const std::array<float, 3> c2,
    const bool add_triangles
) {
    int nodes = 3;
    int classes = 2;
    std::vector<int> src = {0, 0, 1};
    std::vector<int> dest = {1, 2, 2};
    std::vector<float> edge_costs = {edge_cost, edge_cost, edge_cost};
    std::vector<float> class_costs = {
        c1[0], c2[0], c1[1], c2[1], c1[2], c2[2]
    };
    thrust::device_vector<int> i;
    thrust::device_vector<int> j;
    thrust::device_vector<float> costs;
    std::tie(i, j, costs) = mwc_to_coo(nodes, classes, class_costs, src, dest, edge_costs);

    thrust::device_vector<int> t1, t2, t3;
    if (add_triangles) {
        t1 = std::vector<int>{0, 0, 0, 0, 0, 1, 1};
        t2 = std::vector<int>{1, 1, 2, 1, 2, 2, 2};
        t3 = std::vector<int>{2, 3, 3, 4, 4, 3, 4};
    } else {
        t1 = std::vector<int>{};
        t2 = std::vector<int>{};
        t3 = std::vector<int>{};
    }
    dCOO A(i.begin(), i.end(), j.begin(), j.end(), costs.begin(), costs.end(), true);
    multiwaycut_message_passing mwcp(A, nodes, classes, std::move(t1), std::move(t2), std::move(t3));

    const double initial_lb = mwcp.lower_bound();
    std::cout << "initial lb = " << initial_lb << "\n";

    const double expected_initial_lb =
        // Initial lower bound only considers the edge costs
        3 * std::min(edge_cost, 0.0f)  // We have three base edges
        // node-class edges are added to the edges, hence those need to be taken into account as well
        + std::min(c1[0], 0.0f) + std::min(c1[1], 0.0f) + std::min(c1[2], 0.0f)
        + std::min(c2[0], 0.0f) + std::min(c2[1], 0.0f) + std::min(c2[2], 0.0f) ;
    test(std::abs(initial_lb - expected_initial_lb) <= 1e-6, "Initial lb before reparametrization must be " + std::to_string(expected_initial_lb));

    const double expected_final_lb =
        // Edge lower bound, for this test case the class edges should be zero in the end
        3 * std::min(edge_cost, 0.0f)
        // Class lower bound
        + std::min(c1[0], c2[0])
        + std::min(c1[1], c2[1])
        + std::min(c1[2], c2[2]);

    double last_lb = initial_lb;
    for (int k = 0; k < iterations; ++k) {
        std::cout << "---------------" << "iteration = " << k << "---------------\n";
        mwcp.send_messages_to_triplets();
        double new_lb = mwcp.lower_bound();
        test(new_lb > last_lb || std::abs(new_lb - last_lb) < 1e-6, "Lower bound did not increase after message to triplets");
        last_lb = new_lb;

        mwcp.send_messages_to_edges();
        new_lb = mwcp.lower_bound();
        test(new_lb > last_lb || std::abs(new_lb - last_lb) < 1e-6, "Lower bound did not increase after message to edges");
        last_lb = new_lb;

        mwcp.send_messages_from_sum_to_edges();
        new_lb = mwcp.lower_bound();
        test(new_lb > last_lb || std::abs(new_lb - last_lb) < 1e-6, "Lower bound did not increase after messages from class constraints");
        last_lb = new_lb;
    }

    const double final_lb = mwcp.lower_bound();
    std::cout << "final lb = " << final_lb << "\n";

    test(std::abs(final_lb - expected_final_lb) <= 1e-6, "Final lb after reparametrization must be " + std::to_string(expected_final_lb));
}


void test_multiway_cut_repulsive_triangle(
    const float edge_cost,
    const float class_cost,
    const bool add_triangles
) {
    test_multiway_cut_repulsive_triangle(
        edge_cost,
        std::array<float, 3>{{class_cost, class_cost, class_cost}},
        std::array<float, 3>{{class_cost, class_cost, class_cost}},
        add_triangles
    );
}


void test_multiway_cut_2_nodes_2_classes(
    const float edge_cost,
    const std::array<float, 2> c1,  // Costs for class 1, one entry for each node
    const std::array<float, 2> c2,  // Costs for class 2
    const bool add_triangles
) {
    int nodes = 2;
    int classes = 2;
    std::vector<int> src = {0};
    std::vector<int> dest = {1};
    std::vector<float> edge_costs = {edge_cost};
    std::vector<float> class_costs = {
        c1[0], c2[0], c1[1], c2[1]
    };
    thrust::device_vector<int> i;
    thrust::device_vector<int> j;
    thrust::device_vector<float> costs;
    std::tie(i, j, costs) = mwc_to_coo(nodes, classes, class_costs, src, dest, edge_costs);

    thrust::device_vector<int> t1, t2, t3;
    if (add_triangles) {
        t1 = std::vector<int>{0, 0};
        t2 = std::vector<int>{1, 1};
        t3 = std::vector<int>{2, 3};
    } else {
        t1 = std::vector<int>{};
        t2 = std::vector<int>{};
        t3 = std::vector<int>{};
    }


    dCOO A(i.begin(), i.end(), j.begin(), j.end(), costs.begin(), costs.end(), true);
    multiwaycut_message_passing mwcp(A, nodes, classes, std::move(t1), std::move(t2), std::move(t3));

    const double initial_lb = mwcp.lower_bound();
    std::cout << "initial lb = " << initial_lb << "\n";
    const double expected_initial_lb =
        // Initial lower bound only considers the edge costs
        std::min(edge_cost, 0.0f)
        // node-class edges are added to the edges, hence those need to be taken into account as well
        + std::min(c1[0], 0.0f)  + std::min(c1[1], 0.0f)  + std::min(c2[0], 0.0f)  + std::min(c2[1], 0.0f);
    test(std::abs(initial_lb - expected_initial_lb) <= 1e-6, "Initial lb before reparametrization must be " + std::to_string(expected_initial_lb));

    int iterations = 22;  // Need 21 iterations to reach sufficiently close approximation
    double last_lb = initial_lb;
    for (int k = 0; k < iterations; ++k) {
        std::cout << "---------------" << "iteration=" << k << "---------------\n";
        mwcp.send_messages_to_triplets();
        double new_lb = mwcp.lower_bound();
        test(new_lb > last_lb || std::abs(new_lb - last_lb) < 1e-6, "Lower bound did not increase after message to triplets");
        last_lb = new_lb;

        mwcp.send_messages_to_edges();
        new_lb = mwcp.lower_bound();
        test(new_lb > last_lb || std::abs(new_lb - last_lb) < 1e-6, "Lower bound did not increase after message to edges");
        last_lb = new_lb;

        mwcp.send_messages_from_sum_to_edges();
        new_lb = mwcp.lower_bound();
        test(new_lb > last_lb || std::abs(new_lb - last_lb) < 1e-6, "Lower bound did not increase after messages from class constraints");
        last_lb = new_lb;
    }

    const double final_lb = mwcp.lower_bound();
    std::cout << "final lb = " << final_lb << "\n";
    // Lower bound is edge costs + the two smallest class costs
    const double expected_final_lb =
        // Edge lower bound, for this test case the class edges should be set to zero in the end
        std::min(edge_cost, 0.0f)
        // Class lower bound
        + std::min(c1[0], c2[0])
        + std::min(c1[1], c2[1]);
    test(std::abs(final_lb - expected_final_lb) <= 1e-6, "Final lb after reparametrization must be " + std::to_string(expected_final_lb));
}


void test_multiway_cut_2_nodes_2_classes(
    const float edge_cost,
    const float class_cost,
    const bool add_triangles
) {
    test_multiway_cut_2_nodes_2_classes(
        edge_cost,
        std::array<float, 2>{{class_cost, class_cost}},
        std::array<float, 2>{{class_cost, class_cost}},
        add_triangles
    );
}


int main(int argc, char** argv)
{
    std::cout << "Testing repulsive triangle\n";
    test_multiway_cut_repulsive_triangle(-1.0, 0.0, false);
    test_multiway_cut_repulsive_triangle(-1.0, 0.0, true);
    test_multiway_cut_repulsive_triangle(-1.0, -1.0, false);
    test_multiway_cut_repulsive_triangle(-1.0, -1.0, true);
    test_multiway_cut_repulsive_triangle(-1.0, 1.0, false);
    test_multiway_cut_repulsive_triangle(-1.0, 1.0, true);
    std::cout << "Testing 2 nodes 2 classes\n";
    test_multiway_cut_2_nodes_2_classes(1.0, 0.0, false);
    test_multiway_cut_2_nodes_2_classes(1.0, 0.0, true);
    test_multiway_cut_2_nodes_2_classes(1.0, -1.0, false);
    test_multiway_cut_2_nodes_2_classes(1.0, -1.0, true);
    test_multiway_cut_2_nodes_2_classes(1.0, 1.0, false);
    test_multiway_cut_2_nodes_2_classes(1.0, 1.0, true);
}
