#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "dCSR.h"
#include "union_find.hxx"
#include "time_measure_util.h"
#include <algorithm>
#include <cstdlib>

int get_cuda_device()
{
    if(const char* cuda_env = std::getenv("CUDA_VISIBLE_DEVICES"))
    {
        std::cout << "Cuda device number to use = " << std::stoi(cuda_env) << "\n";
        return std::stoi(cuda_env); 
    }
    else
        return 0; 
}

template<typename ITERATOR>
std::tuple<thrust::host_vector<int>, thrust::host_vector<int>, thrust::host_vector<float>> adjacency_edges(ITERATOR entry_begin, ITERATOR entry_end)
{
    const size_t nr_edges = std::distance(entry_begin, entry_end);
    thrust::host_vector<int> col_ids(2*nr_edges);
    thrust::host_vector<int> row_ids(2*nr_edges);
    thrust::host_vector<float> cost(2*nr_edges);
    for(auto it=entry_begin; it!=entry_end; ++it)
    {
        const int i = std::get<0>(*it);
        const int j = std::get<1>(*it);
        const float c = std::get<2>(*it);
        col_ids[2*std::distance(entry_begin, it)] = i;
        row_ids[2*std::distance(entry_begin, it)] = j;
        cost[2*std::distance(entry_begin, it)] = c;
        col_ids[2*std::distance(entry_begin, it)+1] = j;
        row_ids[2*std::distance(entry_begin, it)+1] = i;
        cost[2*std::distance(entry_begin, it)+1] = c;
    }
    return {col_ids, row_ids, cost};
}

std::tuple<dCSR,std::vector<int>> edge_contraction_matrix_cuda(hipsparseHandle_t handle, const std::vector<std::array<int,2>>& edges, const int n)
{
    union_find uf(n);
    for(size_t c=0; c<edges.size(); ++c)
        uf.merge(edges[c][0],edges[c][1]);

    std::vector<char> node_id_present(n,false);
    for(int i=0; i<n; ++i)
        node_id_present[uf.find(i)] = 1;
    std::vector<int> uf_find_mapping(n, std::numeric_limits<int>::max());
    int c=0;
    for(int i=0; i<n; ++i)
        if(node_id_present[i])
            uf_find_mapping[i] = c++;

    assert(c == std::count(node_id_present.begin(), node_id_present.end(), 1));
    std::vector<int> node_mapping;
    node_mapping.reserve(n);
    for(int i=0; i<n; ++i)
    {
        assert(uf_find_mapping[uf.find(i)] != std::numeric_limits<int>::max());
        node_mapping.push_back( uf_find_mapping[uf.find(i)] );
    }

    std::vector<int> col_ids;
    std::vector<int> row_ids;
    std::vector<float> data;
    for(int i=0; i<n; ++i)
    {
        assert(node_mapping[i] < c && node_mapping[i] >= 0);
        row_ids.push_back(i);
        col_ids.push_back(node_mapping[i]);
        data.push_back(1.0);
    }
    dCSR C(handle, col_ids.begin(), col_ids.end(), row_ids.begin(), row_ids.end(), data.begin(), data.end());
    //dCSR C(handle, row_ids.begin(), row_ids.end(), col_ids.begin(), col_ids.end(), data.begin(), data.end());
    std::cout << "edge contraction matrix dim: " << C.cols() << ", " << C.rows() << "\n";

    return {C, node_mapping}; 
}

struct positive_edge_indicator_func
{
    __host__ __device__
        bool operator()(const thrust::tuple<int,int,float> t)
        {
            if(thrust::get<0>(t) > thrust::get<1>(t) && thrust::get<2>(t) > 0.0)
                return false;
            else
                return true;
        }
};

struct edge_comparator_func {
    __host__ __device__
        inline bool operator()(const thrust::tuple<int, int, float>& a, const thrust::tuple<int, int, float>& b)
        {
            return thrust::get<2>(a) < thrust::get<2>(b);
        } 
};

std::tuple<thrust::device_vector<int>, thrust::device_vector<int>> edges_to_contract_cuda(hipsparseHandle_t handle, dCSR& A, const size_t max_contractions)
{
    assert(max_contractions > 0);
    thrust::device_vector<int> col_ids;
    thrust::device_vector<int> row_ids;
    thrust::device_vector<float> data;
    std::tie(col_ids, row_ids, data) = A.export_coo(handle);
    std::cout << "adjacency matrix nr of edges = " << col_ids.size() << "\n";


    auto first = thrust::make_zip_iterator(thrust::make_tuple(col_ids.begin(), row_ids.begin(), data.begin()));
    auto last = thrust::make_zip_iterator(thrust::make_tuple(col_ids.end(), row_ids.end(), data.end()));

    auto new_last = thrust::remove_if(first, last, positive_edge_indicator_func());
    const size_t nr_positive_edges = std::distance(first, new_last);
    col_ids.resize(nr_positive_edges);
    row_ids.resize(nr_positive_edges);
    data.resize(nr_positive_edges);

    if(max_contractions < nr_positive_edges)
    {
        auto first = thrust::make_zip_iterator(thrust::make_tuple(col_ids.begin(), row_ids.begin(), data.begin()));
        auto last = thrust::make_zip_iterator(thrust::make_tuple(col_ids.end(), row_ids.end(), data.end()));
        thrust::sort(first, last, edge_comparator_func()); // TODO: faster through sort by keys?

        col_ids.resize(max_contractions);
        row_ids.resize(max_contractions);
        data.resize(max_contractions);
    }

    return {col_ids, row_ids};
}

std::vector<std::array<int,2>> edges_to_contract(hipsparseHandle_t handle, dCSR& A, const size_t max_contractions)
{
    assert(max_contractions > 0);
    const auto A_coo = A.export_coo(handle);
    const auto& col_ids = std::get<0>(A_coo);
    const auto& row_ids = std::get<1>(A_coo);
    const auto& data = std::get<2>(A_coo);
    std::cout << "adjacency matrix nr of edges = " << col_ids.size() << "\n";
    std::vector<std::tuple<int,int,float>> positive_edges;
    for(size_t c=0; c<col_ids.size(); ++c)
    {
        const int i = col_ids[c];
        const int j = row_ids[c];
        const float x = data[c];
        //std::cout << i << "," << j << "," << x << "\n";
        if(i > j && x > 0.0)
            positive_edges.push_back({i, j, x});
    }
    if(max_contractions < positive_edges.size())
    {
        std::nth_element(positive_edges.begin(), positive_edges.begin() + max_contractions, positive_edges.end(), [](const auto& a, const auto& b) { return std::get<2>(a) > std::get<2>(b); });
        positive_edges.resize(max_contractions);
    }

    std::vector<std::array<int,2>> edge_indices;
    edge_indices.reserve(positive_edges.size());
    for(auto it=positive_edges.begin(); it!=positive_edges.end(); ++it)
    {
        const int i = std::get<0>(*it);
        const int j = std::get<1>(*it);
        edge_indices.push_back({i,j});
    }
    return edge_indices; 
}

std::vector<int> parallel_gaec_cuda(dCSR& A)
{
    hipsparseHandle_t handle;
    checkCuSparseError(hipsparseCreate(&handle), "cusparse init failed");

    const double initial_lb = A.sum()/2.0;
    std::cout << "initial energy = " << initial_lb << "\n";

    std::vector<int> node_mapping(A.rows());
    std::iota(node_mapping.begin(), node_mapping.end(), 0);
    constexpr static double contract_ratio = 0.1;
    assert(A.rows() == A.cols());

    for(size_t iter=0;; ++iter)
    {
        //std::cout << "Adjacency matrix:\n";
        //std::cout << Eigen::MatrixXf(A) << "\n";
        const size_t nr_edges_to_contract = std::max(size_t(1), size_t(A.rows() * contract_ratio));
        
        const auto e = edges_to_contract(handle, A, nr_edges_to_contract);
        std::cout << "edges to contract size = " << e.size() << "\n";
        //std::cout << "iteration " << iter << ", edges to contract = " << e.size() << ", nr nodes remaining = " << A.rows() << "\n";
        if(e.size() == 0)
        {
            std::cout << "# iterations = " << iter << "\n";
            break;
        }
        dCSR C;
        std::vector<int> cur_node_mapping;
        std::tie(C, cur_node_mapping) = edge_contraction_matrix_cuda(handle, e, A.rows());
        for(size_t i=0; i<node_mapping.size(); ++i)
            node_mapping[i] = cur_node_mapping[node_mapping[i]];

        {
            MEASURE_FUNCTION_EXECUTION_TIME;

            assert(A.cols() == A.rows());
            std::cout << "A dim = " << A.cols() << "x" << A.rows() << "\n";
            std::cout << "A*C multiply time:\n";
            dCSR intermed = multiply(handle, A, C);
            std::cout << "C' transpose time:\n";
            dCSR C_trans = C.transpose(handle);
            std::cout << "C' * (AC) multiply time:\n";
            dCSR new_A = multiply(handle, C_trans, intermed);
            std::cout << "C' A C dim = " << new_A.rows() << "x" << new_A.cols() << "\n"; 
            A = new_A;
            assert(A.rows() == A.cols());
            std::cout << "execution time for matrix multiplication:\n";
        }

        A.set_diagonal_to_zero(handle);
    }

    //std::cout << "solution:\n";
    //for(size_t i=0; i<node_mapping.size(); ++i)
    //    std::cout << i << " -> " << node_mapping[i] << "\n";
    const double lb = A.sum()/2.0;
    std::cout << "final energy = " << lb << "\n";

    hipsparseDestroy(handle);
    return node_mapping;
}

std::vector<int> parallel_gaec_cuda(const std::vector<std::tuple<int,int,float>>& edges)
{
    MEASURE_FUNCTION_EXECUTION_TIME;

    const auto adj_edges = adjacency_edges(edges.begin(), edges.end());

    const int cuda_device = get_cuda_device();
    hipSetDevice(cuda_device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device);
    std::cout << "Going to use " << prop.name << " " << prop.major << "." << prop.minor << ", device number " << cuda_device << "\n";
    hipsparseHandle_t handle;
    checkCuSparseError(hipsparseCreate(&handle), "cusparse init failed");

    dCSR A(handle, 
            std::get<0>(adj_edges).begin(), std::get<0>(adj_edges).end(),
            std::get<1>(adj_edges).begin(), std::get<1>(adj_edges).end(),
            std::get<2>(adj_edges).begin(), std::get<2>(adj_edges).end());
    hipsparseDestroy(handle);

    return parallel_gaec_cuda(A); 
}
