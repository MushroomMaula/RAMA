#include "icp_small_cycles.h"
#include <thrust/device_vector.h>
#include <hipsparse.h>
#include "utils.h"
#include "dCOO.h"

int main(int argc, char** argv)
{
    const std::vector<int> i = {0, 1, 0, 2, 3, 0, 2, 0, 3, 4, 5, 4};
    const std::vector<int> j = {1, 2, 2, 3, 4, 3, 4, 4, 5, 5, 6, 6};
    const std::vector<float> costs = {2., 3., -1., 4., 1.5, 5., 2., -2., -3., 2., -1.5, 0.5};

    double lb;
    dCOO A;
    std::tie(lb, A) = parallel_small_cycle_packing_cuda(i, j, costs, 5);
    assert(lb == -2.5);

    hipsparseHandle_t handle;
    checkCuSparseError(hipsparseCreate(&handle), "cusparse init failed");

    // First compute without any packing (re-arranges the edges):
    std::tie(lb, A) = parallel_small_cycle_packing_cuda(i, j, costs, 0);

    // Now, pack cycles:
    dCOO A_packed;
    std::tie(lb, A_packed) = parallel_small_cycle_packing_cuda(i, j, costs, 5);

    thrust::device_vector<float> costs_original_d = A.get_data();
    thrust::device_vector<float> costs_packed_d = A_packed.get_data();

    for (int e = 0; e < A.edges(); e++)
        if (costs_original_d[e] * costs_packed_d[e] < 0)
            std::cout<<"Test failed. Original cost: "<<costs_original_d[e]<<", packed cost: "<<costs_packed_d[e]<<". Signs should match! \n";
}
